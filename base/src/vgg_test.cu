#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <string>

#include "solver.h"

using namespace std;

typedef unsigned char uchar;

int num_train = 128, num_test = 500;

int reverseInt(int n) {
	int bytes = 4;
	unsigned char ch[bytes];
	for (int i = 0; i < bytes; i++) {
		ch[i] = (n >> i * 8) & 255;
	}
	int p = 0;
	for (int i = 0; i < bytes; i++) {
		p += (int) ch[i] << (bytes - i - 1) * 8;
	}
	return p;
}

void readMNIST(vector<vector<uchar> > &train_images, vector<vector<uchar> > &test_images, vector<uchar> &train_labels, vector<uchar> &test_labels) {
	string filename_train_images = "data/train-images.idx3-ubyte";
	string filename_train_labels = "data/train-labels.idx1-ubyte";

	string filename_test_images = "data/t10k-images.idx3-ubyte";
	string filename_test_labels = "data/t10k-labels.idx1-ubyte";

	// read train/test images
	for (int i = 0; i < 2; i++) {
		string filename;
		if (i == 0)
			filename = filename_train_images;
		else
			filename = filename_test_images;

		ifstream f(filename.c_str(), ios::binary);
		if (!f.is_open())
			printf("Cannot read MNIST from %s\n", filename.c_str());

		// read metadata
		int magic_number = 0, n_images = 0, n_rows = 0, n_cols = 0;
		f.read((char *) &magic_number, sizeof(magic_number));
		magic_number = reverseInt(magic_number);
		f.read((char *) &n_images, sizeof(n_images));
		n_images = reverseInt(n_images);
		f.read((char *) &n_rows, sizeof(n_rows));
		n_rows = reverseInt(n_rows);
		f.read((char *) &n_cols, sizeof(n_cols));
		n_cols = reverseInt(n_cols);

		for (int k = 0; k < n_images; k++) {
			vector<uchar> temp;
			temp.reserve(n_rows * n_cols);
			for (int j = 0; j < n_rows * n_cols; j++) {
				uchar t = 0;
				f.read((char *)&t, sizeof(t));
				temp.push_back(t);
			}
			if (i == 0)
				train_images.push_back(temp);
			else
				test_images.push_back(temp);
		}
		f.close();

	}

	// read train/test labels
	for (int i = 0; i < 2; i++) {
		string filename;
		if (i == 0)
			filename = filename_train_labels;
		else
			filename = filename_test_labels;

		ifstream f(filename.c_str(), ios::binary);
		if (!f.is_open())
			printf("Cannot read MNIST from %s\n", filename.c_str());

		// read metadata
		int magic_number = 0, n_labels = 0;
		f.read((char *) &magic_number, sizeof(magic_number));
		magic_number = reverseInt(magic_number);
		f.read((char *) &n_labels, sizeof(n_labels));
		n_labels = reverseInt(n_labels);

		for (int k = 0; k < n_labels; k++) {
			uchar t = 0;
			f.read((char *)&t, sizeof(t));
			if (i == 0)
				train_labels.push_back(t);
			else
				test_labels.push_back(t);
		}

		f.close();

	}
}

void printTimes(vector<float> &time, string filename);

int main(int argc, char *argv[]) {

	
	float *f_train_images, *f_test_images;
	int *f_train_labels, *f_test_labels;
	int rows = 224, cols = 224, channels = 3;
	int input_size = rows * cols * channels;
	// f_train_images = (float *)malloc(num_train * input_size * sizeof(float));
	// f_train_labels = (int *)malloc(num_train * sizeof(int));
	checkCudaErrors(hipHostMalloc(&f_train_images, num_train * input_size * sizeof(float)));
	checkCudaErrors(hipHostMalloc(&f_train_labels, num_train * sizeof(int)));
	f_test_images = (float *)malloc(num_test * input_size * sizeof(float));
	f_test_labels = (int *)malloc(num_test * sizeof(int));

	float *mean_image;
	mean_image = (float *)malloc(input_size * sizeof(float));

	for (int i = 0; i < input_size; i++) {
		mean_image[i] = 0;
		for (int k = 0; k < num_train; k++) {
			mean_image[i] += f_train_images[k * input_size + i];
		}
		mean_image[i] /= num_train;
	}


	for (int i = 0; i < num_train; i++) {
		for (int j = 0; j < input_size; j++) {
			f_train_images[i * input_size + j] -= mean_image[j];
		}
	}

	for (int i = 0; i < num_test; i++) {
		for (int j = 0; j < input_size; j++) {
			f_test_images[i * input_size + j] -= mean_image[j];
		}

	}

	
	// VGG
	vector<LayerSpecifier> layer_specifier;
	{
		ConvDescriptor part0_conv0;
		part0_conv0.initializeValues(3, 64, 3, 3, 224, 224, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part0_conv0;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part0_conv0_actv;
		part0_conv0_actv.initializeValues(RELU, 64, 224, 224);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part0_conv0_actv;
		layer_specifier.push_back(temp);

	}
	{
		ConvDescriptor part0_conv1;
		part0_conv1.initializeValues(64, 64, 3, 3, 224, 224, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part0_conv1;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part0_conv1_actv;
		part0_conv1_actv.initializeValues(RELU, 64, 224, 224);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part0_conv1_actv;
		layer_specifier.push_back(temp);

	}
	{
		PoolingDescriptor pool0;
		pool0.initializeValues(64, 2, 2, 224, 224, 0, 0, 2, 2, POOLING_MAX);
		LayerSpecifier temp;
		temp.initPointer(POOLING);
		*((PoolingDescriptor *)temp.params) = pool0;
		layer_specifier.push_back(temp);
	}
	{
		ConvDescriptor part1_conv0;
		part1_conv0.initializeValues(64, 128, 3, 3, 112, 112, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part1_conv0;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part1_conv0_actv;
		part1_conv0_actv.initializeValues(RELU, 128, 112, 112);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part1_conv0_actv;
		layer_specifier.push_back(temp);

	}
	{
		ConvDescriptor part1_conv1;
		part1_conv1.initializeValues(128, 128, 3, 3, 112, 112, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part1_conv1;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part1_conv1_actv;
		part1_conv1_actv.initializeValues(RELU, 128, 112, 112);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part1_conv1_actv;
		layer_specifier.push_back(temp);

	}
	{
		PoolingDescriptor pool1;
		pool1.initializeValues(128, 2, 2, 112, 112, 0, 0, 2, 2, POOLING_MAX);
		LayerSpecifier temp;
		temp.initPointer(POOLING);
		*((PoolingDescriptor *)temp.params) = pool1;
		layer_specifier.push_back(temp);
	}
	{
		ConvDescriptor part2_conv0;
		part2_conv0.initializeValues(128, 256, 3, 3, 56, 56, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part2_conv0;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part2_conv0_actv;
		part2_conv0_actv.initializeValues(RELU, 256, 56, 56);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part2_conv0_actv;
		layer_specifier.push_back(temp);

	}
	{
		ConvDescriptor part2_conv1;
		part2_conv1.initializeValues(256, 256, 3, 3, 56, 56, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part2_conv1;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part2_conv1_actv;
		part2_conv1_actv.initializeValues(RELU, 256, 56, 56);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part2_conv1_actv;
		layer_specifier.push_back(temp);

	}
	{
		ConvDescriptor part2_conv2;
		part2_conv2.initializeValues(256, 256, 3, 3, 56, 56, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part2_conv2;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part2_conv2_actv;
		part2_conv2_actv.initializeValues(RELU, 256, 56, 56);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part2_conv2_actv;
		layer_specifier.push_back(temp);

	}
	{
		PoolingDescriptor pool2;
		pool2.initializeValues(256, 2, 2, 56, 56, 0, 0, 2, 2, POOLING_MAX);
		LayerSpecifier temp;
		temp.initPointer(POOLING);
		*((PoolingDescriptor *)temp.params) = pool2;
		layer_specifier.push_back(temp);
	}
	{
		ConvDescriptor part3_conv0;
		part3_conv0.initializeValues(256, 512, 3, 3, 28, 28, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part3_conv0;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part3_conv0_actv;
		part3_conv0_actv.initializeValues(RELU, 512, 28, 28);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part3_conv0_actv;
		layer_specifier.push_back(temp);

	}
	{
		ConvDescriptor part3_conv1;
		part3_conv1.initializeValues(512, 512, 3, 3, 28, 28, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part3_conv1;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part3_conv1_actv;
		part3_conv1_actv.initializeValues(RELU, 512, 28, 28);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part3_conv1_actv;
		layer_specifier.push_back(temp);

	}
	{
		ConvDescriptor part3_conv2;
		part3_conv2.initializeValues(512, 512, 3, 3, 28, 28, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part3_conv2;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part3_conv2_actv;
		part3_conv2_actv.initializeValues(RELU, 512, 28, 28);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part3_conv2_actv;
		layer_specifier.push_back(temp);

	}
	{
		PoolingDescriptor pool3;
		pool3.initializeValues(512, 2, 2, 28, 28, 0, 0, 2, 2, POOLING_MAX);
		LayerSpecifier temp;
		temp.initPointer(POOLING);
		*((PoolingDescriptor *)temp.params) = pool3;
		layer_specifier.push_back(temp);
	}
	{
		ConvDescriptor part4_conv0;
		part4_conv0.initializeValues(512, 512, 3, 3, 14, 14, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part4_conv0;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part4_conv0_actv;
		part4_conv0_actv.initializeValues(RELU, 512, 14, 14);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part4_conv0_actv;
		layer_specifier.push_back(temp);

	}
	{
		ConvDescriptor part4_conv1;
		part4_conv1.initializeValues(512, 512, 3, 3, 14, 14, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part4_conv1;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part4_conv1_actv;
		part4_conv1_actv.initializeValues(RELU, 512, 14, 14);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part4_conv1_actv;
		layer_specifier.push_back(temp);

	}
	{
		ConvDescriptor part4_conv2;
		part4_conv2.initializeValues(512, 512, 3, 3, 14, 14, 1, 1, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(CONV);
		*((ConvDescriptor *)temp.params) = part4_conv2;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part4_conv2_actv;
		part4_conv2_actv.initializeValues(RELU, 512, 14, 14);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part4_conv2_actv;
		layer_specifier.push_back(temp);

	}
	{
		PoolingDescriptor pool3;
		pool3.initializeValues(512, 2, 2, 14, 14, 0, 0, 2, 2, POOLING_MAX);
		LayerSpecifier temp;
		temp.initPointer(POOLING);
		*((PoolingDescriptor *)temp.params) = pool3;
		layer_specifier.push_back(temp);
	}
	
	{
		FCDescriptor part5_fc0;
		part5_fc0.initializeValues(7 * 7 * 512, 4096);
		LayerSpecifier temp;
		temp.initPointer(FULLY_CONNECTED);
		*((FCDescriptor *)temp.params) = part5_fc0;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part5_fc0_actv;
		part5_fc0_actv.initializeValues(RELU, 4096, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part5_fc0_actv;
		layer_specifier.push_back(temp);

	}
	{
		FCDescriptor part5_fc1;
		part5_fc1.initializeValues(4096, 4096);
		LayerSpecifier temp;
		temp.initPointer(FULLY_CONNECTED);
		*((FCDescriptor *)temp.params) = part5_fc1;
		layer_specifier.push_back(temp);
	}
	{
		ActivationDescriptor part5_fc1_actv;
		part5_fc1_actv.initializeValues(RELU, 4096, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(ACTV);
		*((ActivationDescriptor *)temp.params) = part5_fc1_actv;
		layer_specifier.push_back(temp);

	}
	{
		FCDescriptor part5_fc2;
		part5_fc2.initializeValues(4096, 1000);
		LayerSpecifier temp;
		temp.initPointer(FULLY_CONNECTED);
		*((FCDescriptor *)temp.params) = part5_fc2;
		layer_specifier.push_back(temp);
	}
	{
		SoftmaxDescriptor s_max;
		s_max.initializeValues(SOFTMAX_ACCURATE, SOFTMAX_MODE_INSTANCE, 1000, 1, 1);
		LayerSpecifier temp;
		temp.initPointer(SOFTMAX);
		*((SoftmaxDescriptor *)temp.params) = s_max;
		layer_specifier.push_back(temp);
	}

	ConvAlgo conv_algo = CONV_ALGO_PERFORMANCE_OPTIMAL;
	string filename("base_p");
	if (argc == 2) {
		if (strcmp(argv[1], "p") == 0) {
			conv_algo = CONV_ALGO_PERFORMANCE_OPTIMAL;
			filename.assign("base_p");
		}
		else if (strcmp(argv[1], "m") == 0) {
			conv_algo = CONV_ALGO_MEMORY_OPTIMAL;
			filename.assign("base_m");
		}
		else {
			printf("invalid argument.. using performance optimal\n");
		}
	}

	int batch_size = 64;
	long long dropout_seed = 1;
	float softmax_eps = 1e-8;
	float init_std_dev = 0.1;

	NeuralNet net(layer_specifier, DATA_FLOAT, batch_size, TENSOR_NCHW, dropout_seed, softmax_eps, init_std_dev, conv_algo);

	int num_epoch = 1000;
	double learning_rate = 1e-3;
	double learning_rate_decay = 0.9;
	
	Solver solver(&net, (void *)f_train_images, f_train_labels, (void *)f_train_images, f_train_labels, num_epoch, SGD, learning_rate, learning_rate_decay, num_train, num_train);
	vector<float> loss;
	vector<float> time;
	solver.getTrainTime(loss, time, 100);
	printTimes(time, filename);

}

void printTimes(vector<float> &time, string filename) {
	float mean_time = 0.0;
	float std_dev = 0.0;
	int N = time.size();
	for (int i = 0; i < N; i++) {
		mean_time += time[i];
	}
	mean_time /= N;
	for (int i = 0; i < N; i++) {
		std_dev += pow(time[i] - mean_time, 2);
	}
	std_dev /= N;
	std_dev = pow(std_dev, 0.5);
	cout << "Average time: " << mean_time << endl;
	cout << "Standard deviation: " << std_dev << endl;

	filename.append(".dat");
	fstream f;
	f.open(filename.c_str(), ios_base::out);

	for (int i = 0; i < N; i++) {
		f << time[i] << endl;
	}
	f << "mean_time: " << mean_time << endl;
	f << "standard_deviation: " << std_dev << endl;
	f.close();

	filename.append(".bin");
	fstream f_bin;
	f_bin.open(filename.c_str(), ios_base::out);
	f_bin.write((char *)&N, sizeof(N));
	for (int i = 0; i < N; i++) {
		f_bin.write((char *)&time[i], sizeof(time[i]));
	}
	f_bin.close();
}

