
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main() {
	hipEvent_t start, stop, done_offload;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&done_offload);
	void *p, *q;
	long size = 1024l * 1024 * 200;
	hipMalloc(&p, size);
	hipHostMalloc(&q, size, hipHostMallocDefault);
	cout << "without split by event\n";
	int N = 100;
	hipEventRecord(start);
	for (int i = 0; i < N; i++) {
		hipMemcpyAsync(q, p, size, hipMemcpyDeviceToHost, NULL);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milli;
	hipEventElapsedTime(&milli, start, stop);
	cout << "Time(ms): " << milli << endl;

	cout << "with split by event\n";
	hipEventRecord(start);
	for (int i = 0; i < N; i++) {
		hipMemcpyAsync(q, p, size, hipMemcpyDeviceToHost, NULL);
		hipEventRecord(done_offload, NULL);
		hipEventSynchronize(done_offload);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);
	cout << "Time(ms): " << milli << endl;

}